#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <cmath>
#include <hiprand/hiprand_kernel.h>

#include "Molecule.h"
#include "Force.h"
#include "util.h"
#include "Grid.h"

__device__ void update_acc(Molecule &particle, Force &force){
    Molecule _particle = particle;
    Force    _force = force;

    _particle.xa = _force.x / _particle.mass;
    _particle.ya = _force.y / _particle.mass;
    _particle.za = _force.z / _particle.mass;

    particle = _particle;
    force    = _force;
}

__device__ int sign(double value){ return value > 0 ? 1 : -1; }

__global__ void calculate_forces(Molecule *particles, size_t num_particles, double time_step,
                                 size_t box_size, double eff_young_modulus, double eff_shear_modulus){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_particles) {
        Molecule _particle = particles[index];
        if (_particle.flag != 0)
        {
            Force _force_i;
            _force_i.x = 0;
            _force_i.y = 0;
            _force_i.z = -4.5;

            for(int i = 0; i < num_particles; i++) {
                Molecule _particle_i = particles[i];
                if (i == index) continue; 
                double x_proj = _particle.x - _particle_i.x;
                double y_proj = _particle.y - _particle_i.y;
                double z_proj = _particle.z - _particle_i.z;

                double dist_sqr = x_proj * x_proj + y_proj * y_proj + z_proj * z_proj;
                double dist = sqrt(dist_sqr);

                double overlap_distance = (_particle.diameter + _particle_i.diameter) / 2.0;
                if (dist < overlap_distance) { // Check if particles are overlapping

                    double overlap = overlap_distance - dist;
                    double sqrt_overlap = sqrt(overlap);
                    //double cube_sqrt_overlap = sqrt_overlap * sqrt_overlap * sqrt_overlap;

                    double normalized_x_proj = x_proj / dist;
                    double normalized_y_proj = y_proj / dist;
                    double normalized_z_proj = z_proj / dist;

                    double eff_radius = (_particle.diameter * _particle_i.diameter) / (2 * (_particle.diameter + _particle_i.diameter));
                    double sqrt_eff_radius = sqrt(eff_radius);

                    //find projections of overlap distance
                    double overlap_normalized_x = overlap * normalized_x_proj;
                    double overlap_normalized_y = overlap * normalized_y_proj;
                    double overlap_normalized_z = overlap * normalized_z_proj;

                    // Calculate damping force components
                    double relative_velocity_x = _particle.xv - _particle_i.xv;
                    double relative_velocity_y = _particle.yv - _particle_i.yv;
                    double relative_velocity_z = _particle.zv - _particle_i.zv;
                    
                    double normal_relative_velocity_x = relative_velocity_x * normalized_x_proj * normalized_x_proj;
                    double normal_relative_velocity_y = relative_velocity_y * normalized_y_proj * normalized_y_proj;
                    double normal_relative_velocity_z = relative_velocity_z * normalized_z_proj * normalized_z_proj;

                    double normal_force_abs = 1.333 * eff_young_modulus * sqrt_eff_radius;

                    double normal_force_x = normal_force_abs * sqrt(overlap_normalized_x * overlap_normalized_x) * sign(x_proj);
                    double normal_force_y = normal_force_abs * sqrt(overlap_normalized_y * overlap_normalized_y) * sign(y_proj);
                    double normal_force_z = normal_force_abs * sqrt(overlap_normalized_z * overlap_normalized_z) * sign(z_proj);
                    
                    double tangential_relative_velocity_x = relative_velocity_x - normal_relative_velocity_x;
                    double tangential_relative_velocity_y = relative_velocity_y - normal_relative_velocity_y;
                    double tangential_relative_velocity_z = relative_velocity_z - normal_relative_velocity_z;

                    double tangential_displacement_x = tangential_relative_velocity_x * time_step;
                    double tangential_displacement_y = tangential_relative_velocity_y * time_step;
                    double tangential_displacement_z = tangential_relative_velocity_z * time_step;

                    double factor_tangential_damping = -8 * eff_shear_modulus * sqrt_eff_radius;
                    double sqrt_factor_tangential_damping = sqrt(factor_tangential_damping);

                    double tangential_force_x = factor_tangential_damping * sqrt(abs(tangential_displacement_x)) * sign(tangential_displacement_x);
                    double tangential_force_y = factor_tangential_damping * sqrt(abs(tangential_displacement_y)) * sign(tangential_displacement_y);
                    double tangential_force_z = factor_tangential_damping * sqrt(abs(tangential_displacement_z)) * sign(tangential_displacement_z);


                    // add damping forces
                    double restitution = 0.8;
                    double log_rest = log(restitution);
                    double sqr_log_rest = log_rest * log_rest;
                    double sum = sqr_log_rest + 9.8696;
                    double beta = - log_rest / sum; 
                    double eff_mass = (_particle.mass * _particle_i.mass) / (_particle.mass + _particle_i.mass);
                    double sqrt_eff_mass = sqrt(eff_mass);
                    double factor_normal_damping = 2 * eff_young_modulus * sqrt_eff_radius;

                    double S_normal_damping_x = factor_normal_damping * sqrt(abs(overlap_normalized_x));
                    double S_normal_damping_y = factor_normal_damping * sqrt(abs(overlap_normalized_y));
                    double S_normal_damping_z = factor_normal_damping * sqrt(abs(overlap_normalized_z));

                    double factor_normal_damping_force = -1.825741858 * beta * sqrt_eff_mass;
                    double normal_damping_force_x = factor_normal_damping_force * sqrt(S_normal_damping_x)  * relative_velocity_x;
                    double normal_damping_force_y = factor_normal_damping_force * sqrt(S_normal_damping_y)  * relative_velocity_y;
                    double normal_damping_force_z = factor_normal_damping_force * sqrt(S_normal_damping_z)  * relative_velocity_z;

                    double factor_tangential_damping_force = -1.825741858 * beta * sqrt(eff_mass);
                    double tangential_damping_force_x = factor_tangential_damping_force * sqrt(abs(factor_tangential_damping) * sqrt(abs(overlap_normalized_x))) * tangential_relative_velocity_x;
                    double tangential_damping_force_y = factor_tangential_damping_force * sqrt(abs(factor_tangential_damping) * sqrt(abs(overlap_normalized_y))) * tangential_relative_velocity_y;
                    double tangential_damping_force_z = factor_tangential_damping_force * sqrt(abs(factor_tangential_damping) * sqrt(abs(overlap_normalized_z))) * tangential_relative_velocity_z;

                    double friction_restriction_x = 1.15 * abs(normal_force_x + normal_damping_force_x);
                    double friction_restriction_y = 1.15 * abs(normal_force_y + normal_damping_force_y);
                    double friction_restriction_z = 1.15 * abs(normal_force_z + normal_damping_force_z);

                    // double total_damping_force_x = tangential_damping_force_x + tangential_force_x < friction_restriction_x ? tangential_damping_force_x + tangential_force_x : friction_restriction_x;
                    // double total_damping_force_y = tangential_damping_force_y + tangential_force_y < friction_restriction_y ? tangential_damping_force_y + tangential_force_y : friction_restriction_y;
                    // double total_damping_force_z = tangential_damping_force_z + tangential_force_z < friction_restriction_z ? tangential_damping_force_z + tangential_force_z : friction_restriction_z;

                    double total_tangential_force_x = abs(tangential_force_x + tangential_damping_force_x) < friction_restriction_x ? tangential_force_x + tangential_damping_force_x : friction_restriction_x * sign(tangential_force_x + tangential_damping_force_x);
                    double total_tangential_force_y = abs(tangential_force_y + tangential_damping_force_x) < friction_restriction_y ? tangential_force_y + tangential_damping_force_y : friction_restriction_y * sign(tangential_force_y + tangential_damping_force_y);
                    double total_tangential_force_z = abs(tangential_force_z + tangential_damping_force_z) < friction_restriction_z ? tangential_force_z + tangential_damping_force_z : friction_restriction_z * sign(tangential_force_z + tangential_damping_force_z);
                    
                    // Update forces on particle
                    _force_i.x += normal_force_x + total_tangential_force_x + normal_damping_force_x;
                    _force_i.y += normal_force_y + total_tangential_force_y + normal_damping_force_y;
                    _force_i.z += normal_force_z + total_tangential_force_z + normal_damping_force_z;
                }
            }
            if(_particle.flag == 1)
            {
            update_acc(particles[index], _force_i); // a(t + 1/2 dt)
            }
        }
    }
}

__global__ void position_update(Molecule *particles, int num_particles, double time_step, size_t box_size) { // spring constant and damping coefficient

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_particles) {
        Molecule _particle = particles[index];
        if (_particle.flag != 0) {
            double time_step_sqr = time_step * time_step;
            double particle_diameter = _particle.diameter;

            // Update particle position
            _particle.x += _particle.xv * time_step + (_particle.xa * time_step_sqr) / 2; // x(t + dt) = x(t) + v(t) * dt + a(t) * dt² / 2
            _particle.y += _particle.yv * time_step + (_particle.ya * time_step_sqr) / 2;
            _particle.z += _particle.zv * time_step + (_particle.za * time_step_sqr) / 2;

            double k = 10.0;
            double b = 0.95;
            // Wall collisions
            // X direction
            if (_particle.x < particle_diameter || _particle.x > box_size - particle_diameter) {
                double overlap = _particle.x < particle_diameter ? particle_diameter - _particle.x : _particle.x - (box_size - particle_diameter);
                double relative_velocity = _particle.xv;
                double spring_force = k * overlap;
                double damping_force = b * relative_velocity;
                double total_force = spring_force - damping_force;
                double acceleration = total_force / _particle.mass; 

                // Update velocity due to collision
                _particle.xv += (_particle.x < particle_diameter ? acceleration : -acceleration) * time_step;
                
                // Reflect position to simulate bounce
                if (_particle.x < 0) {
                    _particle.x = 0;
                } 
                if (_particle.x > box_size)
                {
                    _particle.x = box_size;
                }
            }

            // Y direction
            if (_particle.y < particle_diameter || _particle.y > box_size - particle_diameter) {
                double overlap = _particle.y < particle_diameter ? particle_diameter - _particle.y : _particle.y - (box_size - particle_diameter);
                double relative_velocity = _particle.yv;
                double spring_force = k * overlap;
                double damping_force = b * relative_velocity;
                double total_force = spring_force - damping_force;
                double acceleration = total_force / _particle.mass;

                // Update velocity due to collision
                _particle.yv += (_particle.y < particle_diameter ? acceleration : -acceleration) * time_step;
                
                // Reflect position to simulate bounce
                if (_particle.y < 0) {
                    _particle.y = 0;
                } 
                if (_particle.y > box_size)
                {
                    _particle.y = box_size;
                }
            }

            // Z direction
            if (_particle.z < particle_diameter || _particle.z > box_size - particle_diameter) {
                double overlap = _particle.z < particle_diameter ? particle_diameter - _particle.z : _particle.z - (box_size - particle_diameter);
                double relative_velocity = _particle.zv;
                double spring_force = k * overlap;
                double damping_force = b*relative_velocity;
                double total_force = spring_force - damping_force;
                double acceleration = total_force / _particle.mass;

                // Update velocity due to collision
                _particle.zv += (_particle.z < particle_diameter ? acceleration : -acceleration) * time_step;
                
                // Reflect position to simulate bounce
                if (_particle.z < 0) {
                    _particle.z = 0;
                } 
                if (_particle.z > box_size)
                {
                    _particle.z = box_size;
                }
            }

            // Write updated particle back to global memory
            particles[index] = _particle;
        }
    }
}

__global__ void velocity_update(Molecule *particle, int num_particles,
                                     double time_step, hiprandState *my_curandstate){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_particles){
        Molecule _particle = particle[index];
        if (_particle.flag != 0){
            _particle.xv += _particle.xa * time_step;
            if (abs(_particle.xv) < 0.01) _particle.xv +=  (hiprand_uniform(my_curandstate+index)) - 0.5;
            _particle.yv += _particle.ya * time_step;
            _particle.zv += _particle.za * time_step;

            particle[index] = _particle;
        }
    }
}

__global__ void setup_kernel(hiprandState *state){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

int main(int argc, char *argv[]) {
    // if (argc < 8) {
    //     std::cerr << "Usage: " << argv[0] << " <time_step> <num_steps> <k> <b> <particle_datafile> <cutoff_dist> <box_size>" << std::endl;
    //     return -1;
    // }

    double time_step = atof(argv[1]);
    int num_steps = atoi(argv[2]);
    std::string particle_datafile = argv[3];
    size_t box_size = atoi(argv[4]);
    double poisson_ratio = atof(argv[5]);
    double young_modulus = atof(argv[6]) * pow(10, 4);

    double cell_length_mulltiplier = 3.0;

    double eff_young_modulus = young_modulus / (2 - 2 * poisson_ratio * poisson_ratio);
    double eff_shear_modulus = young_modulus / (4 + 4 * poisson_ratio);
    std::cout << eff_young_modulus << "  " << eff_shear_modulus << "\n";

    size_t num_particles = 0;

    std::ifstream file(particle_datafile);
    if (!file.is_open()) {
        std::cerr << "Error: Unable to open file " << particle_datafile << std::endl;
        return -1;
    }

    std::string line;
    std::getline(file, line);
    num_particles = std::stoi(line);

    size_t size_molecule = num_particles * sizeof(Molecule);

    Molecule *particles;
    hipMallocManaged(&particles, size_molecule);
    if (particles == nullptr) {
        std::cerr << "Failed to allocate managed memory for particles!" << std::endl;
        return -1;
    }

    double max_particle_size = fill_particles(particles, num_particles, cell_length_mulltiplier, box_size, file);

    writeBoxVTK(box_size);  // Write the box boundary to a VTK file

    int NUM_THREAD = 256;
    int NUM_BLOCK = (num_particles + NUM_THREAD - 1) / NUM_THREAD;

    auto start_global = std::chrono::steady_clock::now();
    hipError_t syncErr, asyncErr;

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));
    setup_kernel<<<NUM_BLOCK, NUM_THREAD>>>(d_state);

    for (int i = 0; i < num_steps; i++) {
        //auto start = std::chrono::steady_clock::now();

        position_update<<<NUM_BLOCK, NUM_THREAD>>>(particles, num_particles, time_step, box_size);
        syncErr = hipGetLastError();
        asyncErr = hipDeviceSynchronize();
        if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
        if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

        calculate_forces<<<NUM_BLOCK, NUM_THREAD>>>(particles, num_particles, time_step, box_size, eff_young_modulus, eff_shear_modulus);
        syncErr = hipGetLastError();
        asyncErr = hipDeviceSynchronize();
        if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
        if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

        velocity_update<<<NUM_BLOCK, NUM_THREAD>>>(particles, num_particles, time_step, d_state);
        syncErr = hipGetLastError();
        asyncErr = hipDeviceSynchronize();
        if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
        if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

        if (i % 10 == 0) {
            // auto end = std::chrono::steady_clock::now();
            // auto time = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
            // std::cout << "Iteration took " << time.count() << " milliseconds\n";
            writeVTK(i, num_particles, particles);
        }
    }

    auto end_global = std::chrono::steady_clock::now();
    auto time_global = std::chrono::duration_cast<std::chrono::seconds>(end_global - start_global);
    std::cout << "Total iteration time " << time_global.count() << " seconds\n";

    hipFree(particles);
}



